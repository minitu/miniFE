#include <CudaUtils.h>

namespace miniFE {

  hipStream_t CudaManager::s1;
  hipStream_t CudaManager::s2;
  hipEvent_t CudaManager::e1;
  hipEvent_t CudaManager::e2;
  hipEvent_t CudaManager::et[ET_COUNT];
  bool CudaManager::initialized=false;

}
